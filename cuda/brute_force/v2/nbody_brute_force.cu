#include "hip/hip_runtime.h"
/*
** nbody_brute_force.c - nbody simulation using the brute-force algorithm (O(n*n))
**
**/

#ifdef DISPLAY
#include "utils/ui/ui.h"
#endif

#include "utils/nbody/nbody.h"
#include "utils/nbody/nbody_tools.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

int nparticles = 500; /* number of particles */
float T_FINAL = 1.0;  /* simulation end time */
particle_t *particles;

double sum_speed_sq = 0;
double max_acc = 0;
double max_speed = 0;
double t = 0;

#define MAX_THREADS 1024

// Custom realisations of atomicAdd and atomicMax for doubles if CUDA version < 6.0
// taken from
// https://stackoverflow.com/questions/17399119/cuda-atomicadd-for-double
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions%5B/url%5D

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
static __inline__ __device__ double atomicAdd(double *address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;
  do
  {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

// __device__ __forceinline__ double atomicMax(double* addr, double value) {
//     double old;
//     old = !signbit(value) ? __longlong_as_double(atomicMax((long long int *)addr, __double_as_longlong(value))) :
//         __longlong_as_double(atomicMin((unsigned long long int *)addr, __double_as_longlong(value)));

//     return old;
// }

static __inline__ __device__ double atomicMax(double *address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;

  do
  {
    assumed = old;
    old = atomicCAS(address_as_ull,
                    assumed,
                    __double_as_longlong(max(val, __longlong_as_double(assumed))));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}

extern "C" void init(int *argc, char ***argv)
{
  parse_args(*argc, *argv);

  /* Allocate global shared arrays for the particles data set. */
  particles = (particle_t *)malloc(sizeof(particle_t) * nparticles);
  all_init_particles(nparticles, particles);
}

#if DISPLAY
void draw_particles()
{
  draw_all_particles();
}
#endif

#ifdef DUMP_RESULT

extern "C" void dump_particles(FILE *f)
{
  print_all_particles(f);
}
#endif

__device__ void reset_forces(particle_t *gpu_particles, int n_particles)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= n_particles)
    return;

  particle_t *p = &gpu_particles[i];
  p->x_force = 0;
  p->y_force = 0;
}

__device__ void calculate_forces(particle_t *gpu_particles, int n_particles)
{
  for (int i = blockIdx.x; i < n_particles; i += gridDim.x)
  {
    particle_t *p1 = &gpu_particles[i];

    for (int j = threadIdx.x; j < n_particles; j += blockDim.x)
    {
      particle_t *p2 = &gpu_particles[j];

      double x_sep, y_sep, dist_sq, grav_base;

      x_sep = p2->x_pos - p1->x_pos;
      y_sep = p2->y_pos - p1->y_pos;
      dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

      /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
      grav_base = GRAV_CONSTANT * (p1->mass) * (p2->mass) / dist_sq;

      atomicAdd(&(p1->x_force), grav_base * x_sep);
      atomicAdd(&(p1->y_force), grav_base * y_sep);
    }
  }
}

__device__ void move_all_particles(particle_t *gpu_particles, double step, double *gpu_sum_speed_sq,
                                   double *gpu_max_acc, double *gpu_max_speed, int n_particles)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= n_particles)
    return;

  particle_t *p = &gpu_particles[i];

  p->x_pos += (p->x_vel) * step;
  p->y_pos += (p->y_vel) * step;
  double x_acc = p->x_force / p->mass;
  double y_acc = p->y_force / p->mass;
  p->x_vel += x_acc * step;
  p->y_vel += y_acc * step;

  /* compute statistics */
  double cur_acc = (x_acc * x_acc + y_acc * y_acc);
  cur_acc = sqrt(cur_acc);
  double speed_sq = (p->x_vel) * (p->x_vel) + (p->y_vel) * (p->y_vel);
  double cur_speed = sqrt(speed_sq);

  atomicAdd(gpu_sum_speed_sq, speed_sq);
  atomicMax(gpu_max_acc, cur_acc);
  atomicMax(gpu_max_speed, cur_speed);
}

__global__ void run_simulation_kernel(particle_t *gpu_particles, double *gpu_sum_speed_sq,
                                      double *gpu_max_acc, double *gpu_max_speed,
                                      double *gpu_t, float t_final, int n_particles)
{
  double dt = 0.01;
  auto grid = cooperative_groups::this_grid();

  while ((*gpu_t) < t_final && n_particles > 0)
  {
    /* Update time. */
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
      (*gpu_t) += dt;
    }
    
    reset_forces(gpu_particles, n_particles);

    grid.sync();

    calculate_forces(gpu_particles, n_particles);

    grid.sync();

    /* Move particles with the current and compute rms velocity. */
    move_all_particles(gpu_particles, dt, gpu_sum_speed_sq, gpu_max_acc, gpu_max_speed, n_particles);

    grid.sync();

    dt = 0.1 * (*gpu_max_speed) / (*gpu_max_acc);
  }
}

extern "C" void run_simulation()
{
  // Get the best grid and block size to fit all threads on the GPU at once
  int blockSize, minGridSize;
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, run_simulation_kernel, 0, 0);

  printf("minGridSize = %d, blockSize = %d\n", minGridSize, blockSize);

  // CUDA setup
  particle_t *gpu_particles;
  double *gpu_sum_speed_sq, *gpu_max_acc, *gpu_max_speed, *gpu_t;

  hipMalloc((void **)&gpu_sum_speed_sq, sizeof(double));
  hipMemcpy(&gpu_sum_speed_sq, &sum_speed_sq, sizeof(double), hipMemcpyHostToDevice);

  hipMalloc((void **)&gpu_max_acc, sizeof(double));
  hipMemcpy(&gpu_max_acc, &max_acc, sizeof(double), hipMemcpyHostToDevice);

  hipMalloc((void **)&gpu_max_speed, sizeof(double));
  hipMemcpy(&gpu_max_speed, &max_speed, sizeof(double), hipMemcpyHostToDevice);

  hipMalloc((void **)&gpu_t, sizeof(double));
  hipMemcpy(&gpu_t, &t, sizeof(double), hipMemcpyHostToDevice);

  hipMalloc((void **)&gpu_particles, nparticles * sizeof(particle_t));
  hipMemcpy(gpu_particles, particles, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);

  void *args[] = {&gpu_particles,
                  &gpu_sum_speed_sq,
                  &gpu_max_acc,
                  &gpu_max_speed,
                  &gpu_t,
                  &T_FINAL,
                  &nparticles};

  hipLaunchCooperativeKernel((void *)run_simulation_kernel, minGridSize, blockSize, args);

  hipMemcpy(particles, gpu_particles, nparticles * sizeof(particle_t), hipMemcpyDeviceToHost);
  hipFree(gpu_particles);
  hipFree(gpu_sum_speed_sq);
  hipFree(gpu_max_acc);
  hipFree(gpu_max_speed);
  hipFree(gpu_t);

  hipDeviceSynchronize();
}

// For compatibility with the other implementations
extern "C"
{
  void init_tools(int *argc, char ***argv) {}
  void finalize_tools() {}
  void finalize() {}
  void free_memory() {}
}