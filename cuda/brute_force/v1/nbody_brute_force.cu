#include "hip/hip_runtime.h"
/*
** nbody_brute_force.c - nbody simulation using the brute-force algorithm (O(n*n))
**
**/


#ifdef DISPLAY
#include "utils/ui/ui.h"
#endif

#include "utils/nbody/nbody.h"
#include "utils/nbody/nbody_tools.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int nparticles = 500; /* number of particles */
float T_FINAL = 1.0;  /* simulation end time */
particle_t *particles;

double sum_speed_sq = 0;
double max_acc = 0;
double max_speed = 0;

#define MAX_THREADS 1024

// Custom realisations of atomicAdd and atomicMax for doubles if CUDA version < 6.0
// taken from
// https://stackoverflow.com/questions/17399119/cuda-atomicadd-for-double
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions%5B/url%5D

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
static __inline__ __device__ double atomicAdd(double *address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;
  do
  {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

static __inline__ __device__ double atomicMax(double *address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;

  do
  {
    assumed = old;
    old = atomicCAS(address_as_ull,
                    assumed,
                    __double_as_longlong(max(val, __longlong_as_double(assumed))));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}

extern "C" void init(int argc, char **argv)
{
  parse_args(argc, argv);

  /* Allocate global shared arrays for the particles data set. */
  particles = (particle_t *)malloc(sizeof(particle_t) * nparticles);
  all_init_particles(nparticles, particles);
}

#if DISPLAY
void draw_particles()
{
  draw_all_particles();
}
#endif

#ifdef DUMP_RESULT

extern "C" void dump_particles(FILE *f)
{
  print_all_particles(f);
}
#endif

__global__ void reset_forces(particle_t *gpu_particles)
{
  int i = blockIdx.x;
  gpu_particles[i].x_force = 0;
  gpu_particles[i].y_force = 0;
}

__global__ void calculate_forces(particle_t *gpu_particles)
{
  int i = blockIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  // if j > nparticles, we don't want to do anything
  if (j >= gridDim.x)
    return;

  particle_t *p1 = &gpu_particles[i];
  particle_t *p2 = &gpu_particles[j];

  double x_sep, y_sep, dist_sq, grav_base;

  x_sep = p2->x_pos - p1->x_pos;
  y_sep = p2->y_pos - p1->y_pos;
  dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

  /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
  grav_base = GRAV_CONSTANT * (p1->mass) * (p2->mass) / dist_sq;

  atomicAdd(&(p1->x_force), grav_base * x_sep);
  atomicAdd(&(p1->y_force), grav_base * y_sep);
}

__global__ void move_all_particles(particle_t *gpu_particles, double step, double *gpu_sum_speed_sq,
                                   double *gpu_max_acc, double *gpu_max_speed)
{
  const int i = blockIdx.x;

  particle_t *p = &gpu_particles[i];
  p->x_pos += (p->x_vel) * step;
  p->y_pos += (p->y_vel) * step;
  double x_acc = p->x_force / p->mass;
  double y_acc = p->y_force / p->mass;
  p->x_vel += x_acc * step;
  p->y_vel += y_acc * step;

  /* compute statistics */
  const double cur_acc = sqrt(x_acc * x_acc + y_acc * y_acc);
  const double speed_sq = (p->x_vel) * (p->x_vel) + (p->y_vel) * (p->y_vel);
  const double cur_speed = sqrt(speed_sq);

  atomicAdd(gpu_sum_speed_sq, speed_sq);
  atomicMax(gpu_max_acc, cur_acc);
  atomicMax(gpu_max_speed, cur_speed);
}

// Les kernel sont des points de synchro askip donc ca devrait etre bon
void all_move_particles_kernel(double step, particle_t *gpu_particles, double *gpu_sum_speed_sq,
                               double *gpu_max_acc, double *gpu_max_speed)
{
  reset_forces<<<nparticles, 1>>>(gpu_particles);

  // Since we can't have more than 1024 threads per block, we need to split the blocks
  const int blocks_count = max(1, int(ceil(double(nparticles) / double(MAX_THREADS))));

  calculate_forces<<<dim3(nparticles, blocks_count), dim3(1, MAX_THREADS)>>>(gpu_particles);
  move_all_particles<<<nparticles, 1>>>(gpu_particles, step, gpu_sum_speed_sq, gpu_max_acc, gpu_max_speed);
}

extern "C" void run_simulation()
{
  // CUDA setup
  particle_t *gpu_particles;
  double *gpu_sum_speed_sq, *gpu_max_acc, *gpu_max_speed;

  hipMalloc((void **)&gpu_sum_speed_sq, sizeof(double));
  hipMemcpy(&gpu_sum_speed_sq, &sum_speed_sq, sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void **)&gpu_max_acc, sizeof(double));
  hipMemcpy(&gpu_max_acc, &max_acc, sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void **)&gpu_max_speed, sizeof(double));
  hipMemcpy(&gpu_max_speed, &max_speed, sizeof(double), hipMemcpyHostToDevice);

  hipMalloc((void **)&gpu_particles, nparticles * sizeof(particle_t));
  hipMemcpy(gpu_particles, particles, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);

  double t = 0.0, dt = 0.01;
  while (t < T_FINAL && nparticles > 0)
  {
    /* Update time. */
    t += dt;
    /* Move particles with the current and compute rms velocity. */
    all_move_particles_kernel(dt, gpu_particles, gpu_sum_speed_sq, gpu_max_acc, gpu_max_speed);

    /* Adjust dt based on maximum speed and acceleration--this
       simple rule tries to insure that no velocity will change
       by more than 10% */

    hipMemcpy(&max_speed, gpu_max_speed, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&max_acc, gpu_max_acc, sizeof(double), hipMemcpyDeviceToHost);

    // only for testing
    hipMemcpy(&sum_speed_sq, gpu_sum_speed_sq, sizeof(double), hipMemcpyDeviceToHost);

    dt = 0.1 * max_speed / max_acc;

    //printf("max_speed = %lf, max_acc = %lf, dt = %lf\n", max_speed, max_acc, dt);

    /* Plot the movement of the particle */
#if DISPLAY
    clear_display();
    draw_all_particles();
    flush_display();
#endif
  }

  hipMemcpy(particles, gpu_particles, nparticles * sizeof(particle_t), hipMemcpyDeviceToHost);
  hipFree(gpu_particles);
  hipFree(gpu_sum_speed_sq);
  hipFree(gpu_max_acc);
  hipFree(gpu_max_speed);

  hipDeviceSynchronize();
}

// For compatibility with the other implementations
extern "C"
{
  void init_tools(int argc, char **argv) {}
  void finalize_tools() {}
  void finalize() {}
  void free_memory() {}
}